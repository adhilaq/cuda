#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vecAdd_kernel(float *x, float *y, float *z, float N)
{
    unsigned int i = blockDim.x*blockIdx.x + threadIdx.x;
    if(i < N)
    {
        z[i] = x[i] + y[i];
    }
}

void vecAdd_GPU(float *x, float *y, float *z, float N)
{
    //Allocate GPU Memory
    float *x_d, *y_d, *z_d;
    hipMalloc((void**)&x_d, sizeof(float)*N);
    hipMalloc((void**)&y_d, sizeof(float)*N);
    hipMalloc((void**)&z_d, sizeof(float)*N);

    //Copy to GPU
    hipMemcpy(x_d, x, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, sizeof(float)*N, hipMemcpyHostToDevice);

    //Run the GPU code
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock - 1)/numThreadsPerBlock;
    vecAdd_kernel<<< numBlocks, numThreadsPerBlock >>>(x_d, y_d, z_d, N);

    //Copy from the GPU
    hipMemcpy(z, z_d, sizeof(float)*N, hipMemcpyDeviceToHost);

    //Deallocate GPU Memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}


int main(int argc, char **argv) 
{
    unsigned int N = (argc > 1)?(atoi(argv[1])):(1<<15);

    float* x = (float*) malloc(N*sizeof(float));
    float* y = (float*) malloc(N*sizeof(float));
    float* z = (float*) malloc(N*sizeof(float));

    for(int i=0; i<N; ++i)
    {
        x[i] = rand();
        y[i] = rand();
    }

    vecAdd_GPU(x, y, z, N);

    // Cleanup
    free(x);
    free(y);
    free(z);
    
    return 0;
}
